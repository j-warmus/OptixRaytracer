#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

#define epsilon 0.001f

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );
rtDeclareVariable(float3, attenuation, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute Attribute, );
rtDeclareVariable(float3, normal, attribute Normal, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float1, t, rtIntersectionDistance, );


/*__device__ float3 getBPMult(float3 intersectPos, float3 lightDir, float distanceToLight) {
    Ray shadowRay = make_Ray(intersectPos, lightDir, 1, epsilon, distanceToLight);
    ShadowPayload shadowPayload;
    rtTrace(root, shadowRay, shadowPayload);


    if (shadowPayload.isVisible) {
        float attenuationMult = attenuation.x +
            attenuation.y * distanceToLight +
            attenuation.z * distanceToLight * distanceToLight;

        float diffuseMult = fmax(dot(normal, lightDir), 0);
        float3 lambert = attrib.diffuse * diffuseMult;

        float blinnphongMult = pow(fmax(dot(normal, normalize(-ray.direction + lightDir)), 0), attrib.shininess);
        float3 blinnphong = attrib.specular * blinnphongMult;

        return (lambert + blinnphong) / attenuationMult;
    }
    else {
        return make_float3(0);
    }
}*/

RT_PROGRAM void closestHit()
{
    float3 result = make_float3(0.f,0.f,0.f);
    
    
    float3 intersectPos = ray.origin + t.x * ray.direction;

    float3 lightDir;        // TODO optix::normalize(lightPos - intersectPos)
    float distanceToLight;  // TODO (0 if directional, optix::length(lightPos - intersectPos) if point light)
    //rtPrintf("%i", plights.size());

    // POINT LIGHTS
    for (int i = 0; i < plights.size(); i++) {
        lightDir = normalize(plights[i].position - intersectPos);
        distanceToLight = length(plights[i].position - intersectPos);

        //float3 blinnPhong = getBPMult(intersectPos, lightDir, distanceToLight);


        /*
        *
        *   BLINN-PHONG CODE
        *
        */
        //Ray shadowRay = make_Ray(intersectPos, lightDir, 1, epsilon, distanceToLight);
        //ShadowPayload shadowPayload;
        //rtTrace(root, shadowRay, shadowPayload);
        //if (shadowPayload.isVisible){
            float attenuationMult = attenuation.x +
                attenuation.y * distanceToLight +
                attenuation.z * distanceToLight * distanceToLight;

            float diffuseMult = fmax(dot(normal, lightDir), 0);
            float3 lambert = attrib.diffuse * diffuseMult;

            float blinnphongMult = pow(fmax(dot(normal, normalize(-ray.direction + lightDir)), 0), attrib.shininess);
            float3 blinnphong = attrib.specular * blinnphongMult;


            float3 blinnPhongMult = (lambert + blinnphong) / attenuationMult;
            /*
           *
           *
           *
           */
            result += make_float3(
                plights[i].color.x * blinnPhongMult.x,
                plights[i].color.y * blinnPhongMult.y,
                plights[i].color.z * blinnPhongMult.z
            );
        //}
    }
    





    // DIRECTIONAL
    for (int i = 0; i < dlights.size(); i++) {
        lightDir = normalize(dlights[i].direction);
        distanceToLight = 0;
        //rtPrintf("%f\n",plights[i].color.x);
        //float3 blinnPhong = getBPMult(intersectPos, lightDir, 0); // distance is 0 for directional lights


        /*
        *
        *   BLINN-PHONG CODE
        *
        */
        float attenuationMult = attenuation.x +
            attenuation.y * distanceToLight +
            attenuation.z * distanceToLight * distanceToLight;

        float diffuseMult = fmax(dot(normal, lightDir), 0);
        float3 lambert = attrib.diffuse * diffuseMult;

        float blinnphongMult = pow(fmax(dot(normal, normalize(-ray.direction + lightDir)), 0), attrib.shininess);
        float3 blinnphong = attrib.specular * blinnphongMult;

        float3 blinnPhongMult = (lambert + blinnphong) / attenuationMult;
        /*
       *
       *
       *
       */



        result += make_float3(
            dlights[i].color.x * blinnPhongMult.x,
            dlights[i].color.y * blinnPhongMult.y,
            dlights[i].color.z * blinnPhongMult.z
        );
    }

    payload.radiance = result;
}