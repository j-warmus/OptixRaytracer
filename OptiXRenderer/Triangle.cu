#include "hip/hip_runtime.h"
#include "optix.h"
#include "optix_device.h"
#include "Geometries.h"

using namespace optix;

rtBuffer<Triangle> triangles; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute Attribute, );
rtDeclareVariable(float3, normal, attribute Normal, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and triangle
    Triangle tri = triangles[primIndex];
    float t;

    // Moller-Trumbore: "http://www.graphics.cornell.edu/pubs/1997/MT97.pdf"

    const float EPSILON = 1e-6;
    float3 v0 = tri.vertices[0];
    float3 v1 = tri.vertices[1];
    float3 v2 = tri.vertices[2];
    float3 e1, e2, h, s, q;
    float a, f, u, v;
    e1 = v1 - v0;
    e2 = v2 - v0;
    h = cross(ray.direction, e2);
    a = dot(e1, h);
    if (a > -EPSILON && a < EPSILON) { return; }
    f = 1.f / a;
    s = ray.origin - v0;
    u = f * dot(s, h);
    if (u < 0.f || u > 1.f) { return; }
    q = cross(s, e1);
    v = f * dot(ray.direction, q);
    if (v < 0.f || u + v > 1.f) { return; }
    t = f * dot(e2, q);
    

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes
        attrib = tri.attrs;
        // get and pass normal
        float3 n1 = normalize(cross((v2 - v1), (v1 - v0)));
        normal = (dot(ray.direction, n1) < 0) ? n1 : -1.f*n1;

        rtReportIntersection(0);
    }
    else { return; }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Triangle tri = triangles[primIndex];

    // TODO: implement triangle bouding box
    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;
}