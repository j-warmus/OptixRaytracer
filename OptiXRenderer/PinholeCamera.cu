#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

#include "Payloads.h"

using namespace optix;

rtBuffer<float3, 2> resultBuffer; // used to store the render result

rtDeclareVariable(rtObject, root, , ); // Optix graph

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, ); // a 2d index (x, y)

rtDeclareVariable(int1, frameID, , );

// Camera info 

// TODO:: delcare camera varaibles here

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float, fovx, , );
rtDeclareVariable(float, fovy, , );

RT_PROGRAM void generateRays()
{
    float3 result = make_float3(0.f);

    // I think this is right
    size_t2 screen = resultBuffer.size();

    float2 d;
    d.x = tan(fovx / 2.) * (((launchIndex.x + 0.5) - screen.x / 2.) / (screen.x / 2.));
    d.y = tan(fovy / 2.) * (((launchIndex.y + 0.5) - screen.y / 2.) / (screen.y / 2.));

    float3 dir = normalize(d.x * U + d.y * V + W);
    float epsilon = 0.001f; 

    // TODO: modify the following lines if you need
    // Shoot a ray to compute the color of the current pixel
    Ray ray = make_Ray(eye, dir, 0, epsilon, RT_DEFAULT_MAX);
    Payload payload;
    rtTrace(root, ray, payload);

    // Write the result
    resultBuffer[launchIndex] = payload.radiance;
}