#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

#include "Payloads.h"

using namespace optix;

rtBuffer<float3, 2> resultBuffer; // used to store the render result

rtDeclareVariable(rtObject, root, , ); // Optix graph

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, ); // a 2d index (x, y)

rtDeclareVariable(int1, frameID, , );

// Camera info 

// TODO:: delcare camera varaibles here

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float, fovx, , );
rtDeclareVariable(float, fovy, , );

RT_PROGRAM void generateRays()
{
    float3 result = make_float3(0.f, 0.f, 0.f);

    // I think this is right
    size_t2 screen = resultBuffer.size();

    float2 d;
    d.x = tan(fovx / 2.) * (((launchIndex.x + 0.5) - screen.x / 2.) / (screen.x / 2.));
    d.y = tan(fovy / 2.) * (((launchIndex.y + 0.5) - screen.y / 2.) / (screen.y / 2.));

    float3 origin = eye;
    float3 dir = normalize(d.x * U + d.y * V - W);
    float epsilon = 0.001f; 


    Payload payload;
    payload.origin = make_float3(0.f, 0.f, 0.f);
    payload.dir = make_float3(0.f, 0.f, 0.f);
    payload.radiance = make_float3(0.f, 0.f, 0.f);
    payload.specular = make_float3(0.f, 0.f, 0.f);
    payload.done = false;
    bool depthset = false;
    //payload.done = false;

    bool first_pass = true;

    do {
        // Set max depth in the payload
        if (!depthset) {
            payload.depth = 5;
            depthset = true;
        }
        
        // Trace a ray
        /*if (payload.depth < 5) {
           rtPrintf("depth %d O: %f, %f, %f      D: %f, %f, %f\n", payload.depth, origin.x, origin.y, origin.z, dir.x, dir.y, dir.z);
        }*/
        
        Ray ray = make_Ray(origin, dir, 0, epsilon, RT_DEFAULT_MAX);
        rtTrace(root, ray, payload);

        //if (payload.origin.x +  payload.origin.y + payload.origin.z != 0) 
        //    rtPrintf("depth %d  or %f %f %f pay %f %f %f\n", payload.depth, origin.x, origin.y, origin.z,
        //     payload.origin.x, payload.origin.y, payload.origin.z);

        // Accumulate radiance
        if (first_pass) {
            result += payload.radiance;
            first_pass = false;
        }
        else
        {
            result += payload.radiance * payload.specular;
        }

        // Prepare to shoot next ray
        origin = payload.origin;
        dir = payload.dir;
    } while (!payload.done && payload.depth > 0);


    // Write the result
    resultBuffer[launchIndex] = result;
}