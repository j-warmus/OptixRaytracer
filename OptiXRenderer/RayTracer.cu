#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute Attribute, );
rtDeclareVariable(float3, normal, attribute Normal, );

RT_PROGRAM void closestHit()
{
    // TDOO: calculate the color using the Blinn-Phong reflection model

    float3 result = normal/2 + 0.5;
    payload.radiance = result;
}