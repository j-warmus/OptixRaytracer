#include "hip/hip_runtime.h"
#include "optix.h"
#include "optix_device.h"
#include "Geometries.h"

using namespace optix;

rtBuffer<Triangle> triangles; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and triangle
    Triangle tri = triangles[primIndex];
    float t;

    // Moller-Trumbore: "http://www.graphics.cornell.edu/pubs/1997/MT97.pdf"

    const float EPSILON = 1e-6;
    float3 v0 = tri.vertices[0];
    float3 v1 = tri.vertices[1];
    float3 v2 = tri.vertices[2];
    float3 e1, e2, h, s, q;
    float a, f, u, v;
    e1 = v1 - v0;
    e2 = v2 - v0;
    h = cross(ray.direction, e2);
    a = dot(e1, h);
    if (a > -EPSILON && a < EPSILON) { return; }
    f = 1.f / a;
    s = ray.origin - v0;
    u = f * dot(s, h);
    if (u < 0.f || u > 1.f) { return; }
    q = cross(s, e1);
    v = f * dot(ray.direction, q);
    if (v < 0.f || u + v > 1.f) { return; }
    t = f * dot(e2, q);

    
    // TODO: implement triangle intersection test here

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes

        // TODO: assign attribute variables here

        rtReportIntersection(0);
    }
    else { return; }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Triangle tri = triangles[primIndex];

    // TODO: implement triangle bouding box
    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;
}