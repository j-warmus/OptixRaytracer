#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

#define epsilon 0.001f

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );
rtDeclareVariable(float3, attenuation, , );
rtDeclareVariable(uint, tracedepth, , );
// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute Attribute, );
rtDeclareVariable(float3, normal, attribute Normal, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float1, t, rtIntersectionDistance, );


/*__device__ float3 getBPMult(float3 intersectPos, float3 lightDir, float distanceToLight) {
    Ray shadowRay = make_Ray(intersectPos, lightDir, 1, epsilon, distanceToLight);
    ShadowPayload shadowPayload;
    rtTrace(root, shadowRay, shadowPayload);


    if (shadowPayload.isVisible) {
        float attenuationMult = attenuation.x +
            attenuation.y * distanceToLight +
            attenuation.z * distanceToLight * distanceToLight;

        float diffuseMult = fmax(dot(normal, lightDir), 0);
        float3 lambert = attrib.diffuse * diffuseMult;

        float blinnphongMult = pow(fmax(dot(normal, normalize(-ray.direction + lightDir)), 0), attrib.shininess);
        float3 blinnphong = attrib.specular * blinnphongMult;

        return (lambert + blinnphong) / attenuationMult;
    }
    else {
        return make_float3(0);
    }
}*/

RT_PROGRAM void closestHit()
{
    float3 result = make_float3(0.f, 0.f, 0.f);
    Ray shadowRay;
    ShadowPayload shadowPayload;

    float3 intersectPos = ray.origin + t.x * ray.direction;

    float3 lightDir = make_float3(0, 0, 0);        // TODO optix::normalize(lightPos - intersectPos)
    float distanceToLight = 0;  // TODO (0 if directional, optix::length(lightPos - intersectPos) if point light)

    //rtPrintf("Casting magic spell to make shadows work. %i\n", 1);

    result += attrib.ambient + attrib.emission;

    // POINT LIGHTS
    for (int i = 0; i < plights.size(); i++) {
        lightDir = normalize(plights[i].position - intersectPos);       // surfaces look closer to how they should when not normalized, but center spheres are still weird.
        distanceToLight = length(plights[i].position - intersectPos);

        shadowRay = make_Ray(intersectPos, lightDir, 1, epsilon, distanceToLight);
        shadowPayload.isVisible = true;
        rtTrace(root, shadowRay, shadowPayload);

        if (shadowPayload.isVisible)
        {
            float attenuationMult = attenuation.x +
                attenuation.y * distanceToLight +
                attenuation.z * distanceToLight * distanceToLight;

            float diffuseMult = fmax(dot(normal, lightDir), 0);
            float3 lambert = attrib.diffuse * diffuseMult;

            float blinnphongMult = pow(fmax(dot(normal, normalize(-ray.direction + lightDir)), 0), attrib.shininess);
            float3 blinnphong = attrib.specular * blinnphongMult;


            float3 totalBP = (lambert + blinnphong) / attenuationMult;
            /*
           *
           *
           *
           */
            result += make_float3(
                plights[i].color.x * totalBP.x,
                plights[i].color.y * totalBP.y,
                plights[i].color.z * totalBP.z
            );
        }
    }






    // DIRECTIONAL
    for (int i = 0; i < dlights.size(); i++) {
        lightDir = normalize(dlights[i].direction);
        distanceToLight = 0;
        //rtPrintf("%f\n",plights[i].color.x);
        //float3 blinnPhong = getBPMult(intersectPos, lightDir, 0); // distance is 0 for directional lights


        /*
        *
        *   BLINN-PHONG CODE
        *
        */

        shadowRay = make_Ray(intersectPos, lightDir, 1, epsilon, distanceToLight);
        shadowPayload.isVisible = true;
        rtTrace(root, shadowRay, shadowPayload);

        if (shadowPayload.isVisible)
        {
            float attenuationMult = attenuation.x +
                attenuation.y * distanceToLight +
                attenuation.z * distanceToLight * distanceToLight;

            float diffuseMult = fmax(dot(normal, lightDir), 0);
            float3 lambert = attrib.diffuse * diffuseMult;

            float blinnphongMult = pow(fmax(dot(normal, normalize(-ray.direction + lightDir)), 0), attrib.shininess);
            float3 blinnphong = attrib.specular * blinnphongMult;

            float3 totalBP = (lambert + blinnphong) / attenuationMult;
            /*
           *
           *
           *
           */

            result += make_float3(
                dlights[i].color.x * totalBP.x,
                dlights[i].color.y * totalBP.y,
                dlights[i].color.z * totalBP.z
            );
        }
    }

    // RECURSIVE
    if (payload.depth < tracedepth){
        float3 refDir = normalize(ray.direction + 2 * (dot(-ray.direction, normal)) * normal);
        float3 refPos = intersectPos + epsilon * refDir;
        payload.depth += 1;
        
        Ray refRay = make_Ray(refPos, refDir, 0, epsilon, RT_DEFAULT_MAX);
        rtTrace(root, refRay, payload);
        
        // Accumulate radiance
        result += attrib.specular * payload.radiance;
        
    }

    payload.radiance = result;
}