#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include "Geometries.h"

#ifndef SQRT3f
#define SQRT3f      1.73205080756887729352
#endif

using namespace optix;

rtBuffer<Sphere> spheres; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute Attribute, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and sphere
    Sphere sphere = spheres[primIndex];
    float t;

    optix::Matrix4x4 transinv = sphere.transforms.inverse();

    // pre-transform the ray
    float3 src = optix::make_float3(transinv * optix::make_float4(ray.origin, 1));
    float3 dir = optix::normalize(optix::make_matrix3x3(transinv) * ray.direction);
    float3 eyetocenter = src - sphere.center;

    float a = optix::dot(dir, dir);
    float b = 2 * optix::dot(dir, eyetocenter);
    float c = optix::dot(eyetocenter, eyetocenter) - (sphere.radius * sphere.radius);

    float disc = b * b - 4 * a * c;

    if (disc < 0) {             // NO HIT
        return;
    }
    else if (disc == 0) {       // 1 HIT
        t = (-b / 2 * a);
    }
    else {                      // 2 HITS
        disc = sqrt(disc);
        float sol1 = ((-b + disc) / 2 * a);
        float sol2 = ((-b - disc) / 2 * a);

        if (sol1 == sol2) {
            t = sol1;
        }
        else if (sol1 > 0 && sol2 > 0) {
            t = sol1 < sol2 ? sol1 : sol2;
        }
        else if ((sol1 < 0 && sol2 > 0) || (sol1 > 0 && sol2 < 0)) {
            t = sol1 < 0 ? sol2 : sol1;
        }
        else {
            return;
        }
    }

    // get the location of the (transformed) intersetion
    optix::float3 hitPos = src + t * dir;
    // post-transform the hit position and intersection distance
    hitPos = optix::make_float3(sphere.transforms * optix::make_float4(hitPos, 1));
    t = optix::dot(hitPos - ray.origin, ray.direction);

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes
        attrib = sphere.attrs;
        // rtPrintf("%f\n", t);
        // TODO: assign attribute variables here
        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Sphere sphere = spheres[primIndex];

    // the diagonal of the cube is 2*r*sqrt(3) so the distance from the center to a corner is r*sqrt(3)
    float boundSize = sphere.radius * SQRT3f;

    result[0] = sphere.center.x - boundSize;
    result[1] = sphere.center.y - boundSize;
    result[2] = sphere.center.z - boundSize;
    result[3] = sphere.center.x + boundSize;
    result[4] = sphere.center.y + boundSize;
    result[5] = sphere.center.z + boundSize;
}