#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

#include "Payloads.h"

using namespace optix;

rtBuffer<float3, 2> resultBuffer; // used to store the render result

rtDeclareVariable(rtObject, root, , ); // Optix graph

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, ); // a 2d index (x, y)

rtDeclareVariable(int1, frameID, , );

// Camera info 

// TODO:: delcare camera varaibles here

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );


RT_PROGRAM void generateRays()
{
    float3 result = make_float3(0.f);

    // TODO: calculate the ray direction (change the following lines)
    size_t2 screen = resultBuffer.size();
    float2 d = (make_float2(launchIndex) / make_float2(screen)) * 2.f - 1.f;
    float3 origin = eye;
    float3 dir = normalize(d.x * U + d.y * V + W);
    float epsilon = 0.001f; 

    // TODO: modify the following lines if you need
    // Shoot a ray to compute the color of the current pixel
    Ray ray = make_Ray(origin, dir, 0, epsilon, RT_DEFAULT_MAX);
    Payload payload;
    rtTrace(root, ray, payload);

    // Write the result
    resultBuffer[launchIndex] = payload.radiance;
}