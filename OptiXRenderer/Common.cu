#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>

#include "Payloads.h"

using namespace optix;

rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(float3, backgroundColor, , );
rtDeclareVariable(Ray, ray, rtCurrentRay, );

RT_PROGRAM void miss()
{
    // Set the result to be the background color if miss
    // TODO: change the color to backgroundColor
    // payload.radiance = backgroundColor;
    //if (payload.depth < 5) {
    //    printf("missed\n");
    //}

    payload.radiance = backgroundColor;
    payload.specular = make_float3(1.f, 1.f, 1.f);
     
    if (ray.ray_type == 0) {
        payload.done = true;
    }
}

RT_PROGRAM void exception()
{
    // Print any exception for debugging
    const unsigned int code = rtGetExceptionCode();
    rtPrintExceptionDetails();
}

rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );
rtDeclareVariable(float1, t, rtIntersectionDistance, );

RT_PROGRAM void anyHit()
{
    shadowPayload.isVisible = false;
    rtTerminateRay();
}