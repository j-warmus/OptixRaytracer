#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include "Geometries.h"

using namespace optix;

rtBuffer<Sphere> spheres; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute Attribute, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and sphere
    Sphere sphere = spheres[primIndex];
    float t;

    // TODO: implement sphere intersection test here

    optix::Matrix4x4 transinv = sphere.transforms.inverse();

    float3 src = optix::make_float3(transinv * optix::make_float4(ray.origin, 1));
    float3 dir = optix::normalize(optix::make_matrix3x3(transinv) * ray.direction);
    float3 eyetocenter = src - sphere.center;

    float a = optix::dot(dir, dir);
    float b = 2 * optix::dot(dir, eyetocenter);
    float c = optix::dot(eyetocenter, eyetocenter) - (sphere.radius * sphere.radius);

    float disc = b * b - 4 * a * c;

    if (disc < 0) {             // NO HIT
        return;
    }
    else if (disc == 0) {       // 1 HIT
        t = (-b / 2 * a);
    }
    else {                      // 2 HITS
        disc = sqrt(disc);
        float sol1 = ((-b + disc) / 2 * a);
        float sol2 = ((-b - disc) / 2 * a);

        if (sol1 == sol2) {
            t = sol1;
        }
        else if (sol1 > 0 && sol2 > 0) {
            t = sol1 < sol2 ? sol1 : sol2;
        }
        else if ((sol1 < 0 && sol2 > 0) || (sol1 > 0 && sol2 < 0)) {
            t = sol1 < 0 ? sol2 : sol1;
        }
        else {
            return;
        }
    }


    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes
        attrib = sphere.attrs;
        // rtPrintf("%f\n", t);
        // TODO: assign attribute variables here
        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Sphere sphere = spheres[primIndex];

    // TODO: implement sphere bouding box
    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;
}