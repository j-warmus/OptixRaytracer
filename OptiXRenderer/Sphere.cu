#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include "Geometries.h"

using namespace optix;

rtBuffer<Sphere> spheres; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute Attribute, );
rtDeclareVariable(float3, normal, attribute Normal, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and sphere
    Sphere sphere = spheres[primIndex];
    float t;

    optix::Matrix4x4 transinv = sphere.transforms.inverse();

    // pre-transform the ray
    float3 src = optix::make_float3(transinv * optix::make_float4(ray.origin, 1));
    float3 dir = optix::normalize(optix::make_matrix3x3(transinv) * ray.direction);
    float3 eyetocenter = src - sphere.center;

    float a = optix::dot(dir, dir);
    float b = 2 * optix::dot(dir, eyetocenter);
    float c = optix::dot(eyetocenter, eyetocenter) - (sphere.radius * sphere.radius);

    float disc = b * b - 4 * a * c;

    if (disc < 0) {             // NO HIT
        return;
    }
    else if (disc == 0) {       // 1 HIT
        t = (-b / 2 * a);
    }
    else {                      // 2 HITS
        disc = sqrt(disc);
        float sol1 = ((-b + disc) / 2 * a);
        float sol2 = ((-b - disc) / 2 * a);

        if (sol1 == sol2) {
            t = sol1;
        }
        else if (sol1 > 0 && sol2 > 0) {
            t = sol1 < sol2 ? sol1 : sol2;
        }
        else if ((sol1 < 0 && sol2 > 0) || (sol1 > 0 && sol2 < 0)) {
            t = sol1 < 0 ? sol2 : sol1;
        }
        else {
            return;
        }
    }

    // get the location of the (transformed) intersetion
    optix::float3 hitPos = src + t * dir;
    // post-transform the hit position and intersection distance
    hitPos = optix::make_float3(sphere.transforms * optix::make_float4(hitPos, 1));
    t = optix::dot(hitPos - ray.origin, ray.direction);

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes
        attrib = sphere.attrs;
        // calc and pass normal
        normal = hitPos - make_float3(sphere.transforms * make_float4(sphere.center,1));
        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Sphere sphere = spheres[primIndex];

    // TODO: implement sphere bouding box
    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;
}