#include "hip/hip_runtime.h"
#include "optix.h"
#include "optix_device.h"
#include "Geometries.h"

using namespace optix;

rtBuffer<Triangle> triangles; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute Attribute, );
rtDeclareVariable(float3, normal, attribute Normal, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and triangle
    Triangle tri = triangles[primIndex];
    float t;

    // Moller-Trumbore: "http://www.graphics.cornell.edu/pubs/1997/MT97.pdf"

    const float EPSILON = 1e-6;
    float3 v0 = tri.vertices[0];
    float3 v1 = tri.vertices[1];
    float3 v2 = tri.vertices[2];
    float3 e1, e2, h, s, q;
    float a, f, u, v;
    e1 = v1 - v0;
    e2 = v2 - v0;
    h = cross(ray.direction, e2);
    a = dot(e1, h);
    if (a > -EPSILON && a < EPSILON) { return; }
    f = 1.f / a;
    s = ray.origin - v0;
    u = f * dot(s, h);
    if (u < 0.f || u > 1.f) { return; }
    q = cross(s, e1);
    v = f * dot(ray.direction, q);
    if (v < 0.f || u + v > 1.f) { return; }
    t = f * dot(e2, q);
    

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes
        attrib = tri.attrs;
        // get and pass normal
        float3 n1 = normalize(cross((v2 - v1), (v1 - v0)));
        normal = (dot(ray.direction, n1) < 0) ? n1 : -1.f*n1;

        rtReportIntersection(0);
    }
    else { return; }
}

// helper function for determining the bounding box coordinates
float min3(float a, float b, float c) {
    return a < b ? (a < c ? a : c) : (b < c ? b : c);
}
float max3(float a, float b, float c) {
    return a > b ? (a > c ? a : c) : (b > c ? b : c);
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Triangle tri = triangles[primIndex];

    // just get the minimum/maximum x, y, z from the triangle and set those values to be the bounds.
    result[0] = min3(tri.vertices[0].x, tri.vertices[1].x, tri.vertices[2].x);
    result[1] = min3(tri.vertices[0].y, tri.vertices[1].y, tri.vertices[2].y);
    result[2] = min3(tri.vertices[0].z, tri.vertices[1].z, tri.vertices[2].z);
    result[3] = max3(tri.vertices[0].x, tri.vertices[1].x, tri.vertices[2].x);
    result[4] = max3(tri.vertices[0].y, tri.vertices[1].y, tri.vertices[2].y);
    result[5] = max3(tri.vertices[0].z, tri.vertices[1].z, tri.vertices[2].z);
}